#include "hip/hip_runtime.h"
//	File        :   YnLayerCropGpu.c
//	Brief       :   Implement methods.
//	DD-MM-YYYY  :   15-08-2016
//	Author      :   haittt

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "../include/YnLayerCropGpu.h"
#include "../include/YnCudaGpu.h"
#include "../include/YnBlasGpu.h"
#include "../include/YnImageGpu.h"
}

/**************** Define */

/**************** Macro */

/**************** Enum */

/**************** Struct */

/**************** Local variables */

/**************** Global variables */

/**************** Local Implement */

/**************** Implement */
YN_GPU_DEVICE float _YnLayerCropGpuPixelGet(float *image,
        int w,
        int h,
        int x,
        int y,
        int c)
{
    if (x < 0 || x >= w || y < 0 || y >= h)
        return 0;

    return image[x + w * (y + c * h)];
}

YN_GPU_DEVICE float3 _YnLayerCropGpuRgbToHsv(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = (r > g) ? ( (r > b) ? r : b) : ( (g > b) ? g : b);
    float min = (r < g) ? ( (r < b) ? r : b) : ( (g < b) ? g : b);
    float delta = max - min;

    v = max;
    if (max == 0)
    {
        s = 0;
        h = -1;
    }
    else
    {
        s = delta/max;

        if (r == max)
        {
            h = (g - b) / delta;
        }
        else if (g == max)
        {
            h = 2 + (b - r) / delta;
        }
        else
        {
            h = 4 + (r - g) / delta;
        }

        if (h < 0)
            h += 6;
    }

    return make_float3(h, s, v);
}

YN_GPU_DEVICE float3 _YnLayerCropGpuHsvToRgb(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;
    int index;
    float r, g, b;
    float f, p, q, t;

    if (s == 0)
    {
        r = g = b = v;
    }
    else
    {
        index = (int) floorf(h);
        f = h - index;
        p = v*(1-s);
        q = v*(1-s*f);
        t = v*(1-s*(1-f));

        if (index == 0)
        {
            r = v;
            g = t;
            b = p;
        }
        else if (index == 1)
        {
            r = q;
            g = v;
            b = p;
        }
        else if (index == 2)
        {
            r = p;
            g = v;
            b = t;
        }
        else if (index == 3)
        {
            r = p;
            g = q;
            b = v;
        }
        else if (index == 4)
        {
            r = t;
            g = p;
            b = v;
        }
        else
        {
            r = v;
            g = p;
            b = q;
        }
    }
    r = (r < 0) ? 0 : ((r > 1) ? 1 : r);
    g = (g < 0) ? 0 : ((g > 1) ? 1 : g);
    b = (b < 0) ? 0 : ((b > 1) ? 1 : b);

    return make_float3(r, g, b);
}

YN_GPU_DEVICE float _YnLayerCropGpuBilinearInterpolate(float *image,
        int w,
        int h,
        float x,
        float y,
        int c)
{
    int ix = (int) floorf(x);
    int iy = (int) floorf(y);

    float dx = x - ix;
    float dy = y - iy;

    float val = (1-dy) * (1-dx) * _YnLayerCropGpuPixelGet(image, w, h, ix,     iy,     c) +
                dy     * (1-dx) * _YnLayerCropGpuPixelGet(image, w, h, ix,     iy + 1, c) +
                (1-dy) *   dx   * _YnLayerCropGpuPixelGet(image, w, h, ix + 1, iy,     c) +
                dy     *   dx   * _YnLayerCropGpuPixelGet(image, w, h, ix + 1, iy+1,   c);

    return val;
}

YN_GPU_GLOBAL void  _YnLayerCropGpuLevelsImage(float *image,
        float *rand,
        int batch,
        int w,
        int h,
        int train,
        float saturation,
        float exposure,
        float translate,
        float scale,
        float shift)
{
    uint32 offset;
    int x, y;
    float rshift;
    float gshift;
    float bshift;
    float r0;
    float r1;
    float r2;
    float r3;
    float3 rgb;
    float3 hsv;
    float r;
    float g;
    float b;
    int size = batch * w * h;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size)
        return;

    x = id % w;
    id /= w;
    y = id % h;
    id /= h;
    rshift = rand[0];
    gshift = rand[1];
    bshift = rand[2];
    r0 = rand[8 * id + 0];
    r1 = rand[8 * id + 1];
    r2 = rand[8 * id + 2];
    r3 = rand[8 * id + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > .5) ? 1./saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > .5) ? 1. / exposure : exposure;

    offset = id * h * w * 3;
    image += offset;
    r = image[x + w * (y + h * 0)];
    g = image[x + w * (y + h * 1)];
    b = image[x + w * (y + h * 2)];

    rgb = make_float3(r,g,b);

    if (train)
    {
        hsv = _YnLayerCropGpuRgbToHsv(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = _YnLayerCropGpuHsvToRgb(hsv);
    }
    else
    {
        shift = 0;
    }

    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - .5) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - .5) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - .5) * shift;
}

YN_GPU_GLOBAL void _YnLayerCropGpuForward(float *input,
        float *rand,
        int size,
        int c,
        int h,
        int w,
        int crop_height,
        int crop_width,
        int train,
        int flip,
        float angle,
        float *output)
{
    int i, j;
    int count;
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size)
        return;

    float cx = w/2.;
    float cy = h/2.;

    int count = id;
    j = id % crop_width;
    id /= crop_width;
    i = id % crop_height;
    id /= crop_height;
    int k = id % c;
    id /= c;
    int b = id;

    float r4 = rand[8*b + 4];
    float r5 = rand[8*b + 5];
    float r6 = rand[8*b + 6];
    float r7 = rand[8*b + 7];

    float dw = (w - crop_width)*r4;
    float dh = (h - crop_height)*r5;
    flip = (flip && (r6 > .5));
    angle = 2*angle*r7 - angle;
    if (!train){
        dw = (w - crop_width)/2.;
        dh = (h - crop_height)/2.;
        flip = 0;
        angle = 0;
    }

    input += w*h*c*b;

    float x = (flip) ? w - dw - j - 1 : j + dw;
    float y = i + dh;

    float rx = cos(angle)*(x-cx) - sin(angle)*(y-cy) + cx;
    float ry = sin(angle)*(x-cx) + cos(angle)*(y-cy) + cy;

    output[count] = _YnLayerCropBilineraInterpolate(input, w, h, rx, ry, k);
}

YN_EXTERN_C
void YnLayerCropGpuForward(tYnLayer * layer,
        tYnNetworkState netState)
{
    cuda_random(layer.rand_gpu, layer.batch*8);

    float radians = layer.angle*3.14159265/180.;

    float scale = 2;
    float translate = -1;
    if (layer.noadjust){
        scale = 1;
        translate = 0;
    }

    int size = layer.batch * layer.w * layer.h;

    levels_image_kernel<<<YnCudaGridSize(size), YN_GPU_NUM_THREADS_IN_BLOCK>>>(state.input, layer.rand_gpu, layer.batch, layer.w, layer.h, state.train, layer.saturation, layer.exposure, translate, scale, layer.shift);
    check_error(hipPeekAtLastError());

    size = layer.batch*layer.c*layer.out_w*layer.out_h;

    forward_crop_layer_kernel<<<YnCudaGridSize(size), YN_GPU_NUM_THREADS_IN_BLOCK>>>(state.input, layer.rand_gpu, size, layer.c, layer.h, layer.w, layer.out_h, layer.out_w, state.train, layer.flip, radians, layer.output_gpu);
    check_error(hipPeekAtLastError());
}
