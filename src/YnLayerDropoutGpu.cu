//	File        :   YnLayerDropoutayerGpu.cu
//	Brief       :   Implement methods.
//	DD-MM-YYYY  :   28-08-2016
//	Author      :   haittt

#include "../include/YnLayerDropoutGpu.h"

/**************** Define */

/**************** Macro */

/**************** Enum */

/**************** Struct */

/**************** Local variables */

/**************** Global variables */

/**************** Local Implement */

/**************** Implement */
YN_GPU_GLOBAL void _YnDropout(float *input,
        int size,
        float *rand,
        float prob,
        float scale)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    if(id < size)
        input[id] = (rand[id] < prob) ? 0 : input[id] * scale;
}

void YnLayerDropoutGpuForward(tYnLayer layer,
        tYnNetworkState netState)
{
    if (!state.train) return;
    int size = layer.inputs*layer.batch;
    cuda_random(layer.rand_gpu, size);
    /*
    int i;
    for(i = 0; i < size; ++i){
        layer.rand[i] = rand_uniform();
    }
    cuda_push_array(layer.rand_gpu, layer.rand, size);
    */

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(state.input, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}

void YnLayerDropoutGpuBackward(tYnLayer layer,
        tYnNetworkState netState)
{
    if(!state.delta) return;
    int size = layer.inputs*layer.batch;

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(state.delta, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}
